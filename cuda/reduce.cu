#include <iostream>

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>

//#include <cstdint>

//#include "hip/hip_runtime.h"

#include "mcub/cub/device/device_reduce.cuh"

#define check(err) if(err != hipSuccess){ \
      const char* s; \
      hipDrvGetErrorString(err, &s); \
      cerr << "CUDARuntime error: " << s << endl; \
      assert(false); \
    }

using namespace std;
using namespace hipcub;

namespace{

  template<class T>
  class Data{
  public:
    Data(size_t size)
    : size(size){
      d_temp_storage = NULL;
      temp_storage_bytes = 0;

      hipError_t err = hipMalloc(&out, sizeof(T));
      check(err);

      d_out = (T*)out;

      DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, size);
      
      err = hipMalloc(&temp, sizeof(T) * temp_storage_bytes);
      check(err);

      d_temp = (T*)temp;

    }

    void run(hipDeviceptr_t ptr, size_t size){
      d_in = (T*)ptr;
      DeviceReduce::Sum(d_temp, temp_storage_bytes, d_in, d_out, size);
    }

    void copyOut(void* resultPtr){
      hipError_t err = hipMemcpyDtoH(resultPtr, (hipDeviceptr_t)d_out, sizeof(T));
      check(err);
    }

    T* d_in;
    T* d_out;
    T* d_temp;
    
    hipDeviceptr_t out;
    hipDeviceptr_t temp;

    void            *d_temp_storage;
    size_t          temp_storage_bytes;
    size_t          size;

  };

  Data<double>* _data = NULL;

  template<class T>
  void sum_(hipDeviceptr_t ptr, size_t size, void* resultPtr){
    if(!_data){
      _data = new Data<double>(size);
    }
    _data->run(ptr, size);
    _data->copyOut(resultPtr);
  }

  template<class T>
  void product_(hipDeviceptr_t ptr, size_t size, void* resultPtr){
    /*
    thrust::device_ptr<T> begin((T*)ptr);
    thrust::device_ptr<T> end((T*)ptr + size);

    T result = thrust::reduce(begin, end, T(1), thrust::multiplies<T>());
    memcpy(resultPtr, &result, sizeof(T));
    */
  }

} // namespace

namespace ideas{

void reduce(hipDeviceptr_t ptr,
            size_t size,
            size_t scalarBytes,
            bool isFloat,
            bool isSigned,
            bool isSum,
            void* resultPtr){
  switch(scalarBytes){
    case 8:
      if(isFloat){
        if(isSum){
          sum_<double>(ptr, size, resultPtr);
        }
        else{
          product_<double>(ptr, size, resultPtr);
        }
      }
      else{
        if(isSigned){
          if(isSum){
            sum_<int64_t>(ptr, size, resultPtr);
          }
          else{
            product_<int64_t>(ptr, size, resultPtr);
          }
        }
        else{
          if(isSum){
            sum_<uint64_t>(ptr, size, resultPtr);
          }
          else{
            product_<uint64_t>(ptr, size, resultPtr);
          }
        }
      }
      break;
    case 4:
      if(isFloat){
        if(isSum){
          sum_<float>(ptr, size, resultPtr);
        }
        else{
          product_<float>(ptr, size, resultPtr);
        }
      }
      else{
        if(isSigned){
          if(isSum){
            sum_<int32_t>(ptr, size, resultPtr);
          }
          else{
            product_<int32_t>(ptr, size, resultPtr);
          }
        }
        else{
          if(isSum){
            sum_<uint32_t>(ptr, size, resultPtr);
          }
          else{
            product_<uint32_t>(ptr, size, resultPtr);
          }
        }
      }
      break;
    case 2:
      if(isFloat){
        assert(false);
      }
      else{
        if(isSigned){
          if(isSum){
            sum_<int16_t>(ptr, size, resultPtr);
          }
          else{
            product_<int16_t>(ptr, size, resultPtr);
          }
        }
        else{
          if(isSum){
            sum_<uint16_t>(ptr, size, resultPtr);
          }
          else{
            product_<uint16_t>(ptr, size, resultPtr);
          }
        }
      }
      break;
    case 1:
      if(isFloat){
        assert(false);
      }
      else{
        if(isSigned){
          if(isSum){
            sum_<int8_t>(ptr, size, resultPtr);
          }
          else{
            product_<int8_t>(ptr, size, resultPtr);
          }
        }
        else{
          if(isSum){
            sum_<uint8_t>(ptr, size, resultPtr);
          }
          else{
            product_<uint8_t>(ptr, size, resultPtr);
          }
        }
      }
      break;
    default:
      assert(false);
  }
}

} // namespace ideas
