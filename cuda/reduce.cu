#include <iostream>

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>

//#include <cstdint>

//#include "hip/hip_runtime.h"

#include "mcub/device/device_reduce.cuh"

#define check(err) if(err != hipSuccess){ \
      const char* s; \
      hipDrvGetErrorString(err, &s); \
      cerr << "CUDARuntime error: " << s << endl; \
      assert(false); \
    }

using namespace std;
using namespace hipcub;

extern "C" __device__ void run(int index, void* args, void* result);

typedef void(*KernelFunc)(int, void*, void*);

namespace{

  KernelFunc getKernel(size_t kernel){
    switch(kernel){
      case 0:
        return run;
      default:
        assert(false);
    }

    return NULL;
  }

  template<class T>
  class Reduce{
  public:
    Reduce(size_t kernel, size_t size, void* args)
    : size_(size),
    tempStorage_(NULL),
    tempStorageBytes_(0),
    args_(args){

      kernelFunc_ = getKernel(kernel);

      hipError_t err = hipMalloc(&result_, sizeof(T));
      check(err);

      T* in = NULL;
      T* temp = NULL;
      T* result = NULL;

      DeviceReduce::Sum(temp, tempStorageBytes_,
                        in, result, size_, kernelFunc_, args_);
      
      err = hipMalloc(&temp_, sizeof(T) * tempStorageBytes_);
      check(err);
    }

    void run(hipDeviceptr_t ptr, void* resultPtr){
      T* in = (T*)ptr;
      T* temp = (T*)temp_;
      T* result = (T*)result_;

      DeviceReduce::Sum(temp, tempStorageBytes_,
                        in, result, size_, kernelFunc_, args_);

      hipError_t err = hipMemcpyDtoH(resultPtr, result_, sizeof(T));
      check(err);
    }
    
    hipDeviceptr_t result_;
    hipDeviceptr_t temp_;

    void* tempStorage_;
    size_t tempStorageBytes_;
    size_t size_;
    KernelFunc kernelFunc_;
    void* args_;
  };

  template<class T>
  void sum_(size_t kernel,
            hipDeviceptr_t ptr,
            size_t size,
            void* args,
            void* resultPtr){
    Reduce<T>* r = new Reduce<T>(kernel, size, args);
    r->run(ptr, resultPtr);
  }

  template<class T>
  void product_(size_t kernel,
                hipDeviceptr_t ptr,
                size_t size,
                void* args,
                void* resultPtr){
    assert(false);
  }

} // namespace

namespace ideas{

void reduce(size_t kernel,
            hipDeviceptr_t ptr,
            size_t size,
            size_t scalarBytes,
            bool isFloat,
            bool isSigned,
            bool isSum,
            void* args,
            void* resultPtr){
  switch(scalarBytes){
    case 8:
      if(isFloat){
        if(isSum){
          sum_<double>(kernel, ptr, size, args, resultPtr);
        }
        else{
          product_<double>(kernel, ptr, size, args, resultPtr);
        }
      }
      else{
        if(isSigned){
          if(isSum){
            sum_<int64_t>(kernel, ptr, size, args, resultPtr);
          }
          else{
            product_<int64_t>(kernel, ptr, size, args, resultPtr);
          }
        }
        else{
          if(isSum){
            sum_<uint64_t>(kernel, ptr, size, args, resultPtr);
          }
          else{
            product_<uint64_t>(kernel, ptr, size, args, resultPtr);
          }
        }
      }
      break;
    case 4:
      if(isFloat){
        if(isSum){
          sum_<float>(kernel, ptr, size, args, resultPtr);
        }
        else{
          product_<float>(kernel, ptr, size, args, resultPtr);
        }
      }
      else{
        if(isSigned){
          if(isSum){
            sum_<int32_t>(kernel, ptr, size, args, resultPtr);
          }
          else{
            product_<int32_t>(kernel, ptr, size, args, resultPtr);
          }
        }
        else{
          if(isSum){
            sum_<uint32_t>(kernel, ptr, size, args, resultPtr);
          }
          else{
            product_<uint32_t>(kernel, ptr, size, args, resultPtr);
          }
        }
      }
      break;
    case 2:
      if(isFloat){
        assert(false);
      }
      else{
        if(isSigned){
          if(isSum){
            sum_<int16_t>(kernel, ptr, size, args, resultPtr);
          }
          else{
            product_<int16_t>(kernel, ptr, size, args, resultPtr);
          }
        }
        else{
          if(isSum){
            sum_<uint16_t>(kernel, ptr, size, args, resultPtr);
          }
          else{
            product_<uint16_t>(kernel, ptr, size, args, resultPtr);
          }
        }
      }
      break;
    default:
      assert(false);
  }
}

} // namespace ideas
