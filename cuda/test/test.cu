#include <hip/hip_runtime.h>

#include <iostream>
#include <cassert>

#define check(err) if(err != hipSuccess){ \
      const char* s; \
      hipDrvGetErrorString(err, &s); \
      cerr << "CUDARuntime error: " << s << endl; \
      assert(false); \
    }

using namespace std;

namespace ideas{

  extern void reduce(size_t kernel,
              hipDeviceptr_t ptr,
              size_t size,
              size_t scalarBytes,
              bool isFloat,
              bool isSigned,
              bool isSum,
              void* args,
              void* resultPtr);
  
} // namespace ideas

int main(int argc, char** argv){  
  hipError_t err = hipInit(0);
  check(err);
  
  hipDevice_t device;
  hipCtx_t context;

  err = hipDeviceGet(&device, 0);
  check(err);

  err = hipCtxCreate(&context, 0, device);
  check(err);

  hipDeviceptr_t ptr;

  double r = 0;

  ideas::reduce(0, ptr, 1024, 8, true, true, true, NULL, &r);

  cout << "r = " << r << endl;

  return 0;
}
